
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CACHE_SIZE 128
// Analysis kernels

__device__ int row_idx = 0;

extern "C" __global__ void analysis_lower(uint n_rows, uint *max_lvl, volatile bool *analyzed_rows, volatile uint *row_levels, uint *rows, uint *cols) {
    int row = atomicAdd(&row_idx, 1);
    if (row >= n_rows)
        return;

    uint row_start = rows[row];
    uint row_end = rows[row + 1] - 1;

    uint col;
    uint row_lvl = 0; // We determine to which level this row is going to be added
    for (uint i=row_start; i<row_end; i++) {
        col = cols[i];
        while (!analyzed_rows[col])
            continue;
        uint col_lvl = row_levels[col];
        if (row_lvl <= col_lvl)
            row_lvl = col_lvl + 1;
    }

    atomicMax(max_lvl, row_lvl);
    row_levels[row] = row_lvl;
    analyzed_rows[row] = true;
    // Wrap up
    if (row == n_rows - 1)
        row_idx = 0;
}

extern "C" __global__ void analysis_upper(uint n_rows, uint *max_lvl, volatile bool *analyzed_rows, volatile uint *row_levels, uint *rows, uint *cols) {
    int row = n_rows - 1 - atomicAdd(&row_idx, 1);
    if (row < 0)
        return;

    uint row_start = rows[row];
    uint row_end = rows[row + 1] - 1;

    uint col;
    uint row_lvl = 0;
    for (uint i=row_end; i>row_start; i--) {
        col = cols[i];
        while (!analyzed_rows[col])
            continue;
        uint col_lvl = row_levels[col];
        if (row_lvl <= col_lvl)
            row_lvl = col_lvl + 1;
    }

    atomicMax(max_lvl, row_lvl);
    row_levels[row] = row_lvl;
    analyzed_rows[row] = true;
    // Wrap up
    if (row == 0)
        row_idx = 0;
}

// Solve kernels


template<typename Float>
__device__ void solve_lower(uint nrhs, uint nrows, uint *stack_id, uint *levels, volatile bool *solved_rows, uint* rows, uint* columns, Float* values, volatile Float* x) {

    __shared__ uint lvl_idx;
    __shared__ uint cols_cache[CACHE_SIZE];
    __shared__ Float vals_cache[CACHE_SIZE];

    int thread_idx = threadIdx.x;
    // The current block solves the row at index *stack_id in levels
    if (thread_idx == 0) {
        lvl_idx = atomicAdd(stack_id, 1);
    }
    __syncthreads();

    if (lvl_idx >= nrows)
        return;

    uint row = levels[lvl_idx];
    uint row_start = rows[row];
    uint row_end = rows[row + 1] - 1;
    Float diag_entry = values[row_end];
    Float r;
    if (thread_idx < nrhs)
        r = x[thread_idx * nrows + row];
    uint col;
    Float val;
    for (int i=row_start; i<row_end; ++i) {
        uint cache_idx = (i-row_start) % CACHE_SIZE;
        if (cache_idx == 0) {
            // Update the cache
            if (i + thread_idx < (int)row_end) {
                cols_cache[thread_idx] = columns[i + thread_idx];
                vals_cache[thread_idx] = values[i + thread_idx];
            }
            __syncthreads();
        }

        if (thread_idx < nrhs) {
            // Read current column and corresponding entry in the cache
            col = cols_cache[cache_idx];
            val = vals_cache[cache_idx];
        }
        // Busy wait for the corresponding entry in x to be solved
        if (thread_idx == 0) {
            while (!solved_rows[col])
                continue;
        }
        __syncthreads();

        if (thread_idx < nrhs)
            r -= val * x[thread_idx * nrows + col];

    }

    // Write the final value
    if (thread_idx < nrhs)
        x[thread_idx * nrows + row] = r / diag_entry;

    // Make sure we write all entries before signaling other blocks
    __threadfence();
    __syncthreads();

    if (thread_idx != 0)
        return;

    // Signal other blocks that this entry is available
    solved_rows[row] = true;
}

template<typename Float>
__device__ void solve_upper(uint nrhs, uint nrows, uint *stack_id, uint *levels, volatile bool *solved_rows, uint* rows, uint* columns, Float* values, volatile Float* x) {

    __shared__ uint lvl_idx;
    __shared__ uint cols_cache[CACHE_SIZE];
    __shared__ Float vals_cache[CACHE_SIZE];

    int thread_idx = threadIdx.x;
    // The current block solves the row at index *stack_id in levels
    if (thread_idx == 0)
        lvl_idx = atomicAdd(stack_id, 1);
    __syncthreads();

    if (lvl_idx >= nrows)
        return;

    uint row = levels[lvl_idx];
    uint row_start = rows[row];
    uint row_end = rows[row + 1] - 1;
    Float diag_entry = values[row_start];
    Float r;
    if (thread_idx < nrhs)
        r = x[thread_idx * nrows + row];
    uint col;
    Float val;
    for (int i=row_end; i>row_start; --i) {
        int cache_idx = (row_end - i) % CACHE_SIZE;
        if (cache_idx == 0) {
            // Update the cache
            if (i - thread_idx > (int)row_start) {
                vals_cache[thread_idx] = values[i - thread_idx];
                cols_cache[thread_idx] = columns[i - thread_idx];
            }
            __syncthreads();
        }

        if (thread_idx < nrhs) {
            // Read current column and corresponding entry in the cache
            col = cols_cache[cache_idx];
            val = vals_cache[cache_idx];
        }
        // Busy wait for the corresponding entry in x to be solved
        if (thread_idx == 0) {
            while (!solved_rows[col])
                continue;
        }
        __syncthreads();

        if (thread_idx < nrhs)
            r -= val * x[thread_idx * nrows + col];

    }

    // Write the final value
    if (thread_idx < nrhs)
        x[thread_idx * nrows + row] = r / diag_entry;


    // Make sure we write all entries before signaling other blocks
    __threadfence();
    __syncthreads();

    if (thread_idx != 0)
        return;

    // Signal other blocks that this entry is available
    solved_rows[row] = true;
}

extern "C" __global__ void solve_lower_float(uint nrhs, uint nrows, uint *stack_id,  uint *levels, bool *solved_rows, uint *rows, uint *columns, float *values, float*x) {
    solve_lower<float>(nrhs, nrows, stack_id, levels, solved_rows, rows, columns, values, x);
}

extern "C" __global__ void solve_lower_double(uint nrhs, uint nrows, uint *stack_id, uint *levels, bool *solved_rows, uint *rows, uint *columns, double *values, double*x) {
    solve_lower<double>(nrhs, nrows, stack_id, levels, solved_rows, rows, columns, values, x);
}

extern "C" __global__ void solve_upper_float(uint nrhs, uint nrows, uint *stack_id, uint *levels, bool *solved_rows, uint *rows, uint *columns, float *values, float*x) {
    solve_upper<float>(nrhs, nrows, stack_id, levels, solved_rows, rows, columns, values, x);
}

extern "C" __global__ void solve_upper_double(uint nrhs, uint nrows, uint *stack_id, uint *levels, bool *solved_rows, uint *rows, uint *columns, double *values, double*x) {
    solve_upper<double>(nrhs, nrows, stack_id, levels, solved_rows, rows, columns, values, x);
}
